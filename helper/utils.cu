#include "utils.h"

void print2DData(FILE *output, double *data, long width, long height)
{
	long k = 0;
	for (int i = 0; i < height; i++)
	{
		for (int j = 0; j < width; j++)
		{
			fprintf(output, " %2.1f\t", data[k++]);
		}
		fprintf(output, "\n");
	}
	fprintf(output, "\n");
}

void print2DData(FILE *output, float *data, long width, long height)
{
	long k = 0;
	for (int i = 0; i < height; i++)
	{
		for (int j = 0; j < width; j++)
		{
			fprintf(output, " %2.1f\t", data[k++]);
		}
		fprintf(output, "\n");
	}
	fprintf(output, "\n");
}

TIME_USEC get_current_usec()
{
	struct timeval time;
	gettimeofday(&time, NULL);

	TIME_USEC msec;
	msec = (time.tv_sec * 1000000);
	msec += (time.tv_usec);
	return msec;
}
//
//double measure_nsec(struct timespec start){
//	struct timespec end;
//	clock_gettime(CLOCK_MONOTONIC, &end);
//	return end.tv_sec - start.tv_sec + (end.tv_nsec - start.tv_nsec) / 1000000000.;
//}

char *getArgumentValue(int argc, char **argv, char *argName)
{
	char *result = NULL;

	for (int i = 0; i < argc; i++)
	{
		char *substr = strstr(argv[i], argName);
		if (substr != NULL)
		{
			result = strtok(substr, "=");
			if (result != NULL)
			{
				result = strtok(NULL, "=");
			}
			break;
		}
	}

	return result;
}

int msleep(unsigned long milisec)
{
	struct timespec req = {0};
	time_t sec = (int)(milisec / 1000);
	milisec = milisec - (sec * 1000);
	req.tv_sec = sec;
	req.tv_nsec = milisec * 1000000L;
	while (nanosleep(&req, &req) == -1)
		continue;
	return 1;
}

long safeSize(long size)
{
	if (size < SAFE_SIZE)
		size = SAFE_SIZE;

	return size;
}

void setIntValue(int *lvalue, char *rvalue, int defaultValue)
{
	if (rvalue != NULL)
	{
		*lvalue = atoi(rvalue);
	}
	//	else{
	//		*lvalue = defaultValue;
	//	}
}

void setFloatValue(float *lvalue, char *rvalue, float defaultValue)
{
	if (rvalue != NULL)
	{
		*lvalue = atof(rvalue);
	}
	//	else{
	//		*lvalue = defaultValue;
	//	}
}

void parseParams(int argc, char *argv[])
{
	char *paramsFileName = NULL;

	// This is the case no parameters given. In this case it will read values
	// default params file
	if (argc > 1)
	{
		char tempstr[] = "pf";
		paramsFileName = getArgumentValue(argc, argv, tempstr);
	}

	if (paramsFileName != NULL)
	{
		ifstream file(paramsFileName);
		string str;
		argv = new char *[20]();
		char tempstr[] = "";

		argv[0] = tempstr;
		argc = 1;

		while (std::getline(file, str))
		{
			const char *lineArray = str.c_str();
			if (strlen(lineArray) == 0)
				continue;

			char firstChar = lineArray[0];
			if (firstChar != '\t' && firstChar != '#')
			{
				argv[argc] = new char[30]();
				strcpy(argv[argc], str.c_str());
				argc++;
			}
		}
		file.close();
	}
}

void HandleError(hipError_t err, const char *file, int line)
{
	if (err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			   file, line);
		exit(EXIT_FAILURE);
	}
}
